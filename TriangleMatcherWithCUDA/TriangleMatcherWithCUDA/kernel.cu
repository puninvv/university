#include "hip/hip_runtime.h"
#include "hip/hip_runtime.h"
#include ""

#include <malloc.h>
#include <stdlib.h>
#include <time.h>
#include <math.h>

#define defaultRow() blockIdx.y*blockDim.y + threadIdx.y
#define defaultColumn() blockIdx.x*blockDim.x + threadIdx.x
#define ceilMod(x ,y) (x + y - 1)/(y)

#define PI 3.141592f
const int defaultThreadCount = 16;

typedef struct { float x; float y; } Point;
typedef struct { Point A; Point B; Point C; } Triangle;
typedef struct { float dx; float dy; float sin_phi; float cos_phi; } Transformation;
typedef struct { Transformation transformation; float distance; } TransformationWithDistance;

//�����
__host__ float countDistanceBetweenPointsHOST(Point first, Point second)
{
	return (first.x - second.x)*(first.x - second.x) + (first.y - second.y)*(first.y - second.y);
}
__host__ Point countMovedPointHOST(Point p, float dx, float dy)
{
	Point result;
	result.x = p.x + dx;
	result.y = p.y + dy;
	return result;
}
__host__ Point countRotatedPointHOST(Point p, float cos_phi, float sin_phi)
{
	Point result;
	result.x = p.x * cos_phi - p.y * sin_phi;
	result.y = p.x * sin_phi + p.y * cos_phi;
	return result;
}

//������������
__host__ float countDistanceBetweenTrianglesHOST(Triangle* first, Triangle* second)
{
	return
		countDistanceBetweenPointsHOST(first->A, second->A) +
		countDistanceBetweenPointsHOST(first->B, second->B) +
		countDistanceBetweenPointsHOST(first->C, second->C);
}
__host__ Point countTriangleMassCenterHOST(Triangle* ABC)
{
	Point result;
	result.x = (ABC->A.x + ABC->B.x + ABC->C.x) / 3;
	result.y = (ABC->A.y + ABC->B.y + ABC->C.y) / 3;
	return result;
}
__host__ Triangle countMovedTriangleHOST(Triangle* ABC, float dx, float dy)
{
	Triangle result;
	result.A = countMovedPointHOST(ABC->A, dx, dy);
	result.B = countMovedPointHOST(ABC->B, dx, dy);
	result.C = countMovedPointHOST(ABC->C, dx, dy);
	return result;
}
__host__ Triangle countRotatedTriangleHOST(Triangle* ABC, float cos_phi, float sin_phi)
{
	Triangle result;
	result.A = countRotatedPointHOST(ABC->A, cos_phi, sin_phi);
	result.B = countRotatedPointHOST(ABC->B, cos_phi, sin_phi);
	result.C = countRotatedPointHOST(ABC->C, cos_phi, sin_phi);
	return result;
}
__host__ Triangle countTransformedTriangleHOST(Triangle* ABC, Transformation t)
{
	Point ABCmc = countTriangleMassCenterHOST(ABC);
	Triangle ABC_moved = countMovedTriangleHOST(ABC, -ABCmc.x, -ABCmc.y);
	Triangle ABC_moved_rotated = countRotatedTriangleHOST(&ABC_moved, t.cos_phi, t.sin_phi);
	return countMovedTriangleHOST(&ABC_moved_rotated, t.dx, t.dy);
}

//��������������
__host__ float countOptimumlPhiHOST(float phi0, float sProd, float vProd, int maxIterations, float e)
{
	float resultPhi = phi0;

	float cos_phi;
	float sin_phi;
	float newPhi;
	for (int i = 0; i < maxIterations; i++)
	{
		cos_phi = cosf(resultPhi);
		sin_phi = sinf(resultPhi);

		newPhi = resultPhi - (sin_phi * sProd + cos_phi * vProd) / (cos_phi * sProd - sin_phi * vProd);

		if (fabsf(newPhi - resultPhi) < e)
			return newPhi;
		else
			resultPhi = newPhi;
	}

	return resultPhi;
}
__host__ TransformationWithDistance findOptimumTransformationABCHOST(Triangle* ABC_, Triangle* ABC, float e, int maxIterations, int parts)
{
	Point ABCmc = countTriangleMassCenterHOST(ABC);
	Triangle movedABC = countMovedTriangleHOST(ABC, -ABCmc.x, -ABCmc.y);

	Point ABC_mc = countTriangleMassCenterHOST(ABC_);
	Triangle movedABC_ = countMovedTriangleHOST(ABC_, -ABC_mc.x, -ABC_mc.y);

	float sProd = movedABC.A.x * movedABC_.A.x + movedABC.A.y * movedABC_.A.y + movedABC.B.x * movedABC_.B.x + movedABC.B.y * movedABC_.B.y + movedABC.C.x * movedABC_.C.x + movedABC.C.y * movedABC_.C.y;
	float vProd = movedABC.A.x * movedABC_.A.y - movedABC.A.y * movedABC_.A.x + movedABC.B.x * movedABC_.B.y - movedABC.B.y * movedABC_.B.x + movedABC.C.x * movedABC_.C.y - movedABC.C.y * movedABC_.C.x;

	TransformationWithDistance result;
	result.transformation.dx = ABCmc.x; result.transformation.dy = ABCmc.y;
	result.transformation.cos_phi = 1; result.transformation.sin_phi = 0;
	result.distance = countDistanceBetweenTrianglesHOST(&movedABC, &movedABC_);

	float optimumPhi;
	float optimum_cos;
	float optimum_sin;
	float step = 2 * PI / parts;
	Triangle tmpResult;
	float distance;

	for (int i = 0; i < parts; i++)
	{
		optimumPhi = countOptimumlPhiHOST(i * step - PI, sProd, vProd, maxIterations, e);
		optimum_cos = cosf(optimumPhi);
		optimum_sin = sinf(optimumPhi);

		tmpResult = countRotatedTriangleHOST(&movedABC_, optimum_cos, optimum_sin);

		distance = countDistanceBetweenTrianglesHOST(&movedABC, &tmpResult);
		if (distance < result.distance)
		{
			result.distance = distance;
			result.transformation.cos_phi = optimum_cos;
			result.transformation.sin_phi = optimum_sin;
		}
	}

	return result;
}
__host__ TransformationWithDistance findOptimumTransformationHOST(Triangle* ABC_, Triangle* ABC, float e, int maxIterations, int parts)
{
	TransformationWithDistance twdABC = findOptimumTransformationABCHOST(ABC_, ABC, e, maxIterations, parts);

	Triangle tmpTriangle;
	tmpTriangle.A = ABC_->B;
	tmpTriangle.B = ABC_->C;
	tmpTriangle.C = ABC_->A;
	TransformationWithDistance twdBCA = findOptimumTransformationABCHOST(&tmpTriangle, ABC, e, maxIterations, parts);

	tmpTriangle.A = ABC_->C;
	tmpTriangle.B = ABC_->A;
	tmpTriangle.C = ABC_->B;
	TransformationWithDistance twdCAB = findOptimumTransformationABCHOST(&tmpTriangle, ABC, e, maxIterations, parts);

	if (twdBCA.distance < twdABC.distance)
		twdABC = twdBCA;

	return (twdCAB.distance < twdABC.distance) ? twdCAB : twdABC;
}


//�����
__device__ float countDistanceBetweenPointsDEVICE(Point first, Point second)
{
	return (first.x - second.x)*(first.x - second.x) + (first.y - second.y)*(first.y - second.y);
}
__device__ Point countMovedPointDEVICE(Point p, float dx, float dy)
{
	Point result;
	result.x = p.x + dx;
	result.y = p.y + dy;
	return result;
}
__device__ Point countRotatedPointDEVICE(Point p, float cos_phi, float sin_phi)
{
	Point result;
	result.x = p.x * cos_phi - p.y * sin_phi;
	result.y = p.x * sin_phi + p.y * cos_phi;
	return result;
}

//������������
__device__ float countDistanceBetweenTrianglesDEVICE(Triangle* first, Triangle* second)
{
	return
		countDistanceBetweenPointsDEVICE(first->A, second->A) +
		countDistanceBetweenPointsDEVICE(first->B, second->B) +
		countDistanceBetweenPointsDEVICE(first->C, second->C);
}
__device__ Point countTriangleMassCenterDEVICE(Triangle* ABC)
{
	Point result;
	result.x = (ABC->A.x + ABC->B.x + ABC->C.x) / 3;
	result.y = (ABC->A.y + ABC->B.y + ABC->C.y) / 3;
	return result;
}
__device__ Triangle countMovedTriangleDEVICE(Triangle* ABC, float dx, float dy)
{
	Triangle result;
	result.A = countMovedPointDEVICE(ABC->A, dx, dy);
	result.B = countMovedPointDEVICE(ABC->B, dx, dy);
	result.C = countMovedPointDEVICE(ABC->C, dx, dy);
	return result;
}
__device__ Triangle countRotatedTriangleDEVICE(Triangle* ABC, float cos_phi, float sin_phi)
{
	Triangle result;
	result.A = countRotatedPointDEVICE(ABC->A, cos_phi, sin_phi);
	result.B = countRotatedPointDEVICE(ABC->B, cos_phi, sin_phi);
	result.C = countRotatedPointDEVICE(ABC->C, cos_phi, sin_phi);
	return result;
}
__device__ Triangle countTransformedTriangleDEVICE(Triangle* ABC, Transformation t)
{
	Point ABCmc = countTriangleMassCenterDEVICE(ABC);
	Triangle ABC_moved = countMovedTriangleDEVICE(ABC, -ABCmc.x, -ABCmc.y);
	Triangle ABC_moved_rotated = countRotatedTriangleDEVICE(&ABC_moved, t.cos_phi, t.sin_phi);
	return countMovedTriangleDEVICE(&ABC_moved_rotated, t.dx, t.dy);
}

//��������������
__device__ float countOptimumlPhiDEVICE(float phi0, float sProd, float vProd, int maxIterations, float e)
{
	float resultPhi = phi0;

	float cos_phi;
	float sin_phi;
	float newPhi;
	for (int i = 0; i < maxIterations; i++)
	{
		cos_phi = cosf(resultPhi);
		sin_phi = sinf(resultPhi);

		newPhi = resultPhi - (sin_phi * sProd + cos_phi * vProd) / (cos_phi * sProd - sin_phi * vProd);

		if (fabsf(newPhi - resultPhi) < e)
			return newPhi;
		else
			resultPhi = newPhi;
	}

	return resultPhi;
}
__device__ TransformationWithDistance findOptimumTransformationABCDEVICE(Triangle* ABC_, Triangle* ABC, float e, int maxIterations, int parts)
{
	Point ABCmc = countTriangleMassCenterDEVICE(ABC);
	Triangle movedABC = countMovedTriangleDEVICE(ABC, -ABCmc.x, -ABCmc.y);

	Point ABC_mc = countTriangleMassCenterDEVICE(ABC_);
	Triangle movedABC_ = countMovedTriangleDEVICE(ABC_, -ABC_mc.x, -ABC_mc.y);

	float sProd = movedABC.A.x * movedABC_.A.x + movedABC.A.y * movedABC_.A.y + movedABC.B.x * movedABC_.B.x + movedABC.B.y * movedABC_.B.y + movedABC.C.x * movedABC_.C.x + movedABC.C.y * movedABC_.C.y;
	float vProd = movedABC.A.x * movedABC_.A.y - movedABC.A.y * movedABC_.A.x + movedABC.B.x * movedABC_.B.y - movedABC.B.y * movedABC_.B.x + movedABC.C.x * movedABC_.C.y - movedABC.C.y * movedABC_.C.x;

	TransformationWithDistance result;
	result.transformation.dx = ABCmc.x; result.transformation.dy = ABCmc.y;
	result.transformation.cos_phi = 1; result.transformation.sin_phi = 0;
	result.distance = countDistanceBetweenTrianglesDEVICE(&movedABC, &movedABC_);

	float optimumPhi;
	float optimum_cos;
	float optimum_sin;
	float step = 2 * PI / parts;
	Triangle tmpResult;
	float distance;

	for (int i = 0; i < parts; i++)
	{
		optimumPhi = countOptimumlPhiDEVICE(i * step - PI, sProd, vProd, maxIterations, e);
		optimum_cos = cosf(optimumPhi);
		optimum_sin = sinf(optimumPhi);

		tmpResult = countRotatedTriangleDEVICE(&movedABC_, optimum_cos, optimum_sin);

		distance = countDistanceBetweenTrianglesDEVICE(&movedABC, &tmpResult);
		if (distance < result.distance)
		{
			result.distance = distance;
			result.transformation.cos_phi = optimum_cos;
			result.transformation.sin_phi = optimum_sin;
		}
	}

	return result;
}
__device__ TransformationWithDistance findOptimumTransformationDEVICE(Triangle* ABC_, Triangle* ABC, float e, int maxIterations, int parts)
{
	TransformationWithDistance twdABC = findOptimumTransformationABCDEVICE(ABC_, ABC, e, maxIterations, parts);

	Triangle tmpTriangle;
	tmpTriangle.A = ABC_->B;
	tmpTriangle.B = ABC_->C;
	tmpTriangle.C = ABC_->A;
	TransformationWithDistance twdBCA = findOptimumTransformationABCDEVICE(&tmpTriangle, ABC, e, maxIterations, parts);

	tmpTriangle.A = ABC_->C;
	tmpTriangle.B = ABC_->A;
	tmpTriangle.C = ABC_->B;
	TransformationWithDistance twdCAB = findOptimumTransformationABCDEVICE(&tmpTriangle, ABC, e, maxIterations, parts);

	if (twdBCA.distance < twdABC.distance)
		twdABC = twdBCA;

	return (twdCAB.distance < twdABC.distance) ? twdCAB : twdABC;
}


__global__ void fOTKernel(Triangle* ABC_, int ABC_size, Triangle* ABC, int ABCsize, TransformationWithDistance* result, int maxIterations, float e, int parts)
{
	extern __shared__ Triangle cache[];
		
	int row = defaultRow();
	int column = defaultColumn();

	if (row < ABC_size && column < ABCsize)
	{
		Triangle abc_ = ABC_[row];
		Triangle abc = ABC[column];
		result[row * ABCsize + column] = findOptimumTransformationDEVICE(&abc_, &abc, e, maxIterations, parts);
	}
}
hipError_t findOptimumTransformationWithCuda(Triangle* ABC_, int ABC_size, Triangle* ABC, int ABCsize, TransformationWithDistance* result, int maxIterations, float e, int parts)
{
	
	Triangle* devABC_;
	Triangle* devABC;
	TransformationWithDistance* devResult;

	hipError_t cudaStatus;

	cudaStatus = hipMalloc((void**)& devABC_, ABC_size * sizeof(Triangle));
	if (cudaStatus != hipSuccess)
		goto Error;

	cudaStatus = hipMalloc((void**)& devABC, ABCsize * sizeof(Triangle));
	if (cudaStatus != hipSuccess)
		goto Error;

	cudaStatus = hipMalloc((void**)& devResult, ABC_size * ABCsize * sizeof(TransformationWithDistance));
	if (cudaStatus != hipSuccess)
		goto Error;

	cudaStatus = hipMemcpy(devABC_, ABC_, ABC_size * sizeof(Triangle), hipMemcpyHostToDevice);
	if (cudaStatus != hipSuccess)
		goto Error;

	cudaStatus = hipMemcpy(devABC, ABC, ABCsize * sizeof(Triangle), hipMemcpyHostToDevice);
	if (cudaStatus != hipSuccess)
		goto Error;

	dim3 threads(defaultThreadCount, defaultThreadCount);
	dim3 blocks(ceilMod(ABC_size, defaultThreadCount), ceilMod(ABCsize,defaultThreadCount));

	//void findOptimumTransformationKernel(Triangle* ABC_, int ABC_size, Triangle* ABC, int ABCsize, TransformationWithDistance* result, int maxIterations, float e, int parts)
	fOTKernel <<< blocks, threads, 2*sizeof(Triangle)*defaultThreadCount >>>(devABC_, ABC_size, devABC, ABCsize, devResult, maxIterations, e, parts);

	cudaStatus = hipGetLastError();
	if (cudaStatus != hipSuccess)
		goto Error;

	cudaStatus = hipDeviceSynchronize();
	if (cudaStatus != hipSuccess)
		goto Error;

	cudaStatus = hipMemcpy(result, devResult, ABC_size * ABCsize * sizeof(TransformationWithDistance), hipMemcpyDeviceToHost);
	if (cudaStatus != hipSuccess)
		goto Error;

Error:
	hipFree(devABC_);
	hipFree(devABC);
	hipFree(devResult);

	return cudaStatus;
}

int main()
{
	int max_rand = 100;

	int ABCsize = 100;
	int ABC_size = 200;
	Triangle* ABC = (Triangle*)malloc(ABCsize * sizeof(Triangle));
	Triangle* ABC_ = (Triangle*)malloc(ABC_size * sizeof(Triangle));

	srand(time(NULL));                      // ������������� ������� rand ��������� ������� time

	for (int i = 0; i < ABCsize; i++)
	{
		Triangle ABCt;

		ABCt.A.x = max_rand / 2 - rand() % max_rand;
		ABCt.A.y = max_rand / 2 - rand() % max_rand;

		ABCt.B.x = max_rand / 2 - rand() % max_rand;
		ABCt.B.y = max_rand / 2 - rand() % max_rand;

		ABCt.C.x = max_rand / 2 - rand() % max_rand;
		ABCt.C.y = max_rand / 2 - rand() % max_rand;
		ABC[i] = ABCt;
	}

	for (int i = 0; i < ABC_size; i++)
	{
		Triangle ABCt;

		ABCt.A.x = max_rand / 2 - rand() % max_rand;
		ABCt.A.y = max_rand / 2 - rand() % max_rand;

		ABCt.B.x = max_rand / 2 - rand() % max_rand;
		ABCt.B.y = max_rand / 2 - rand() % max_rand;

		ABCt.C.x = max_rand / 2 - rand() % max_rand;
		ABCt.C.y = max_rand / 2 - rand() % max_rand;
		ABC_[i] = ABCt;
	}


	TransformationWithDistance* result = (TransformationWithDistance*)malloc(ABC_size * ABCsize * sizeof(TransformationWithDistance));
	hipError_t cudaStatus = findOptimumTransformationWithCuda(ABC_, ABC_size, ABC, ABCsize, result, 10, 0.00001f, 3);

	if (cudaStatus != hipSuccess)
		goto End;

	cudaStatus = hipDeviceReset();
	if (cudaStatus != hipSuccess)
		goto End;

End:
	free(ABC);
	free(ABC_);
	free(result);

	return 0;
}